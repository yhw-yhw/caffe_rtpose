#include "hip/hip_runtime.h"
#include "caffe/cpm/util/math_functions.hpp"

#define NUMBER_THREADS_PER_BLOCK_1D 32

namespace caffe {

__global__ void fill_image(const float* src_pointer, int w, int h,
                           float* dst_pointer, int boxsize, const float* info, int p) {
  // get pixel location (x,y) within (boxsize, boxsize)
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(x < boxsize && y < boxsize){
    int xr_center = int(info[2*(p+1)] + 0.5);
    int yr_center = int(info[2*(p+1)+1] + 0.5);

    int x_src = xr_center - boxsize/2 + x;
    int y_src = yr_center - boxsize/2 + y;

    int offset_dst = boxsize * boxsize;
    int offset_src = w * h;

    if(x_src >= 0 && x_src < w && y_src >= 0 && y_src < h){
      dst_pointer[                 y * boxsize + x] = src_pointer[                 y_src * w + x_src];
      dst_pointer[offset_dst     + y * boxsize + x] = src_pointer[offset_src     + y_src * w + x_src];
      dst_pointer[offset_dst * 2 + y * boxsize + x] = src_pointer[offset_src * 2 + y_src * w + x_src];
    }
    else {
      dst_pointer[                 y * boxsize + x] = 0;
      dst_pointer[offset_dst     + y * boxsize + x] = 0;
      dst_pointer[offset_dst * 2 + y * boxsize + x] = 0;
    }
  }
}

__global__ void fill_gassian(float* dst_pointer, int boxsize, float sigma){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(x < boxsize && y < boxsize){
    float center_x, center_y;
    center_x = center_y = boxsize / 2;
    float d2 = (x - center_x) * (x - center_x) + (y - center_y) * (y - center_y);
    float exponent = d2 / 2.0 / sigma / sigma;
    if(exponent > 4.6052){ //ln(100) = -ln(1%)
      dst_pointer[y * boxsize + x] = 0;
    }
    else {
      dst_pointer[y * boxsize + x] = exp(-exponent);
    }
  }
}

void fill_pose_net(const float* image, int width, int height,
                   float* dst, int boxsize,
                   const float* peak_pointer_gpu, vector<int> num_people, int limit){
  //image            in width * height * 3 * N
  //dst              in boxsize * boxsize * 4 * (P1+P2+...+PN)
  //peak_pointer_gpu in 2 * 11 * 1 * N
  //num_people has length P, indicating P1, ..., PN
  CHECK(0) << "FIX THIS FUNCTION";
  int N = num_people.size();
  int count = 0;
  bool full = false;
  int offset_src = width * height * 3;
  int offset_dst_2 = boxsize * boxsize;
  int offset_info = 22;
  dim3 threadsPerBlock(NUMBER_THREADS_PER_BLOCK_1D, NUMBER_THREADS_PER_BLOCK_1D);
  dim3 numBlocks(updiv(boxsize, threadsPerBlock.x), updiv(boxsize, threadsPerBlock.y));

  for(int i = 0; i < N; i++){
    //LOG(ERROR) << "copying " << num_people[i] << " people.";
    for(int p = 0; p < num_people[i]; p++){
      fill_image<<<threadsPerBlock, numBlocks>>>(image + i * offset_src, width, height,
                                                 dst + count * (4 * offset_dst_2), boxsize,
                                                 peak_pointer_gpu + i * offset_info, p);
      //src, w, h, dst, boxsize, info, p

      fill_gassian<<<threadsPerBlock, numBlocks>>>(dst + count * (4 * offset_dst_2) + 3 * offset_dst_2, boxsize, 21);
      //dst, boxsize

      count++;
      if(count >= limit){
        full = true;
        break;
      }
    }
    if(full) break;
  }
  hipDeviceSynchronize();
}

}  // namespace caffe
